#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Helper.cuh"

namespace gtom
{
	hipArray_t d_MallocArray(int2 dims)
	{
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipArray_t a_input;
		hipMallocArray(&a_input, &desc, dims.x, dims.y);

		return a_input;
	}

	// Helper function to replace deprecated hipMemcpyToArray with hipMemcpy3D
	void d_CopyToArray(void* d_src, hipArray_t dst, size_t width, size_t height, size_t depth, size_t elemSize, hipMemcpyKind kind)
	{
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr = make_hipPitchedPtr(d_src, width * elemSize, width, height);
		copyParams.dstArray = dst;
		copyParams.extent = make_hipExtent(width, height, depth);
		copyParams.kind = kind;
		hipMemcpy3D(&copyParams);
	}
	
	// Helper function to create a texture object from a CUDA array
	cudaTex d_CreateTextureObject(hipArray_t array, 
								  hipTextureFilterMode filterMode, 
								  hipTextureReadMode readMode, 
								  bool normalizedCoords,
								  hipTextureAddressMode addressMode = hipAddressModeWrap)
	{
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = array;
		
		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.filterMode = filterMode;
		texDesc.readMode = readMode;
		texDesc.normalizedCoords = normalizedCoords;
		texDesc.addressMode[0] = addressMode;
		texDesc.addressMode[1] = addressMode;
		texDesc.addressMode[2] = addressMode;
		
		cudaTex texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
		return texObj;
	}

	void d_MemcpyToArray(tfloat* d_input, hipArray_t a_output, int2 dims)
	{
		// Use the helper function
		d_CopyToArray(d_input, a_output, dims.x, dims.y, 1, sizeof(tfloat), hipMemcpyDeviceToDevice);
	}

	void d_BindTextureToArray(tfloat* d_input, hipArray_t &createdarray, cudaTex &createdtexture, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
	{
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipArray* a_input;
		hipMallocArray(&a_input, &desc, dims.x, dims.y);
		
		// Use our helper function to copy to array
		d_CopyToArray(d_input, a_input, dims.x, dims.y, 1, sizeof(tfloat), hipMemcpyDeviceToDevice);
		
		// Use our helper function to create texture object
		cudaTex texObj = d_CreateTextureObject(a_input, filtermode, hipReadModeElementType, normalizedcoords);

		createdarray = a_input;
		createdtexture = texObj;
	}

	void d_BindTextureToArray(hipArray_t a_input, cudaTex& createdtexture, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
	{
		// Use our helper function to create texture object
		createdtexture = d_CreateTextureObject(a_input, filtermode, hipReadModeElementType, normalizedcoords);
	}

	void d_BindTextureToArray(tfloat* d_input, hipArray_t* &h_createdarrays, cudaTex* &h_createdtextures, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords, int nimages)
	{
		for (int n = 0; n < nimages; n++)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
			hipArray* a_input;
			hipMallocArray(&a_input, &desc, dims.x, dims.y);
			
			// Use our helper function to copy to array
			d_CopyToArray(d_input + Elements2(dims) * n, a_input, dims.x, dims.y, 1, sizeof(tfloat), hipMemcpyDeviceToDevice);
			
			// Use our helper function to create texture object
			cudaTex texObj = d_CreateTextureObject(a_input, filtermode, hipReadModeElementType, normalizedcoords);

			h_createdarrays[n] = a_input;
			h_createdtextures[n] = texObj;
		}
	}

	void d_BindTextureTo3DArray(tfloat* d_input, hipArray_t &createdarray, cudaTex &createdtexture, int3 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
	{
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipArray* a_input;
		hipMalloc3DArray(&a_input, &desc, make_hipExtent(dims.x, dims.y, dims.z));

		hipMemcpy3DParms p = { 0 };
		p.extent = make_hipExtent(dims.x, dims.y, dims.z);
		p.srcPtr = make_hipPitchedPtr(d_input, dims.x * sizeof(tfloat), dims.x, dims.y);
		p.dstArray = a_input;
		p.kind = hipMemcpyDeviceToDevice;
		hipMemcpy3D(&p);

		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = a_input;

		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.filterMode = filtermode;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = normalizedcoords;
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.addressMode[2] = hipAddressModeWrap;
		cudaTex texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

		createdarray = a_input;
		createdtexture = texObj;
	}

	void d_BindTextureTo3DArray(tfloat* d_input, hipArray_t* &h_createdarrays, cudaTex* &h_createdtextures, int3 dims, hipTextureFilterMode filtermode, bool normalizedcoords, int nvolumes)
	{
		for (int n = 0; n < nvolumes; n++)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
			hipArray* a_input;
			hipMalloc3DArray(&a_input, &desc, make_hipExtent(dims.x, dims.y, dims.z));

			hipMemcpy3DParms p = { 0 };
			p.extent = make_hipExtent(dims.x, dims.y, dims.z);
			p.srcPtr = make_hipPitchedPtr(d_input + Elements(dims) * n, dims.x * sizeof(tfloat), dims.x, dims.y);
			p.dstArray = a_input;
			p.kind = hipMemcpyDeviceToDevice;
			hipMemcpy3D(&p);

			struct hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = a_input;

			struct hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.filterMode = filtermode;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = normalizedcoords;
			texDesc.addressMode[0] = hipAddressModeWrap;
			texDesc.addressMode[1] = hipAddressModeWrap;
			texDesc.addressMode[2] = hipAddressModeWrap;
			cudaTex texObj = 0;
			hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

			h_createdarrays[n] = a_input;
			h_createdtextures[n] = texObj;
		}
	}
}