#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Generics.cuh"

namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	template <class T> __global__ void MultiplyByVectorKernel(T* d_input, T* multiplicators, T* d_output, size_t elements);
	template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* d_output, size_t elements, T multiplicator);
	template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* multiplicators, T* d_output, size_t elements);

	__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements);
	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tfloat multiplicator);
	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements);

	__global__ void ComplexDivideByVectorKernel(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements);
	__global__ void ComplexDivideSafeByVectorKernel(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements);

	__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements);
	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator);
	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements);

	__global__ void ComplexMultiplyByConjVectorKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements);
	__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator);
	__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements);

	template <class T> __global__ void DivideByVectorKernel(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch);
	template <class T> __global__ void DivideSafeByVectorKernel(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch);
	template <class T> __global__ void DivideByScalarKernel(T* d_input, T* d_output, size_t elements, T divisor);
	template <class T> __global__ void DivideByScalarKernel(T* d_input, T* d_divisors, T* d_output, size_t elements);

	template <class T> __global__ void AddVectorKernel(T* d_input, T* d_summands, T* d_output, size_t elements, int batch);
	template <class T> __global__ void AddScalarKernel(T* d_input, T* d_output, size_t elements, T summand);
	template <class T> __global__ void AddScalarKernel(T* d_input, T* d_summands, T* d_output, size_t elements);

	template <class T> __global__ void SubtractVectorKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch);
	template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_output, size_t elements, T subtrahend);
	template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements);

	template <class T> __global__ void SquareKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void SqrtKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void PowKernel(T* d_input, T* d_output, size_t elements, T exponent);
	template <class T> __global__ void AbsKernel(T* d_input, T* d_output, size_t elements);
	__global__ void AbsKernel(tcomplex* d_input, tfloat* d_output, size_t elements);
	template <class T> __global__ void InvKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void LogKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void ExpKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void OneMinusKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void SignKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void CosKernel(T* d_input, T* d_output, size_t elements); 
	template <class T> __global__ void SinKernel(T* d_input, T* d_output, size_t elements);
	template <class T> __global__ void MultiplyAddKernel(T* d_mult1, T* d_mult2, T* d_summand, T* d_output, size_t elements);

	__global__ void ComplexPolarToCartKernel(tcomplex* d_polar, tcomplex* d_cart, size_t elements);
	__global__ void ComplexCartToPolarKernel(tcomplex* d_cart, tcomplex* d_polar, size_t elements);
	__global__ void ComplexNormalizeKernel(tcomplex* d_input, tcomplex* d_output, size_t elements);

	template <class T> __global__ void MaxOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements);
	template <class T> __global__ void MaxOpKernel(T* d_input1, T input2, T* d_output, size_t elements);
	template <class T> __global__ void MinOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements);
	template <class T> __global__ void MinOpKernel(T* d_input1, T input2, T* d_output, size_t elements);


	//////////////////
	//Multiplication//
	//////////////////

	template <class T> void d_MultiplyByVector(T* d_input, T* d_multiplicators, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		for (int b = 0; b < batch; b += 32767)
		{
			int curbatch = tmin(32767, batch - b);
			dim3 grid = dim3((uint)totalblocks, tmin(32767, curbatch));
			MultiplyByVectorKernel<T> << <grid, (uint)TpB >> > (d_input + b * elements, d_multiplicators, d_output + b * elements, elements);
		}
	}
	template void d_MultiplyByVector<tfloat>(tfloat* d_input, tfloat* d_multiplicators, tfloat* d_output, size_t elements, int batch);
	template void d_MultiplyByVector<half>(half* d_input, half* d_multiplicators, half* d_output, size_t elements, int batch);
	template void d_MultiplyByVector<int>(int* d_input, int* d_multiplicators, int* d_output, size_t elements, int batch);

	template <class T> void d_MultiplyByScalar(T* d_input, T* d_output, size_t elements, T multiplicator)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		MultiplyByScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements, multiplicator);
	}
	template void d_MultiplyByScalar<half>(half* d_input, half* d_output, size_t elements, half multiplicator);
	template void d_MultiplyByScalar<float>(float* d_input, float* d_output, size_t elements, float multiplicator);
	template void d_MultiplyByScalar<double>(double* d_input, double* d_output, size_t elements, double multiplicator);
	template void d_MultiplyByScalar<int>(int* d_input, int* d_output, size_t elements, int multiplicator);

	template <class T> void d_MultiplyByScalar(T* d_input, T* d_multiplicators, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		MultiplyByScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}
	template void d_MultiplyByScalar<half>(half* d_input, half* d_multiplicators, half* d_output, size_t elements, int batch);
	template void d_MultiplyByScalar<float>(float* d_input, float* d_multiplicators, float* d_output, size_t elements, int batch);
	template void d_MultiplyByScalar<double>(double* d_input, double* d_multiplicators, double* d_output, size_t elements, int batch);
	template void d_MultiplyByScalar<int>(int* d_input, int* d_multiplicators, int* d_output, size_t elements, int batch);

	template <class T> __global__ void MultiplyByVectorKernel(T* d_input, T* d_multiplicators, T* d_output, size_t elements)
	{
		T val;

		size_t offset = elements * blockIdx.y;
		d_output += offset;
		d_input += offset;

		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_multiplicators[id];
			d_output[id] = d_input[id] * val;
		}
	}

	template<> __global__ void MultiplyByVectorKernel<half>(half* d_input, half* d_multiplicators, half* d_output, size_t elements)
	{
		float val;

		size_t offset = elements * blockIdx.y;
		d_output += offset;
		d_input += offset;

		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = __half2float(d_multiplicators[id]);
			d_output[id] = __float2half(__half2float(d_input[id]) * val);
		}
	}

	template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* d_output, size_t elements, T multiplicator)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_input[id] * multiplicator;
	}

	template<> __global__ void MultiplyByScalarKernel<half>(half* d_input, half* d_output, size_t elements, half multiplicator)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = __float2half(__half2float(d_input[id]) * __half2float(multiplicator));
	}

	template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* d_multiplicators, T* d_output, size_t elements)
	{
		T scalar = d_multiplicators[blockIdx.y];

		size_t offset = elements * blockIdx.y;
		d_output += offset;
		d_input += offset;

		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_input[id] * scalar;
	}

	template<> __global__ void MultiplyByScalarKernel<half>(half* d_input, half* d_multiplicators, half* d_output, size_t elements)
	{
		half scalar = d_multiplicators[blockIdx.y];

		size_t offset = elements * blockIdx.y;
		d_output += offset;
		d_input += offset;

		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = __float2half(__half2float(d_input[id]) * __half2float(scalar));
	}


	//////////////////////////
	//Complex Multiplication//
	//////////////////////////

	void d_ComplexMultiplyByVector(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768), batch);
		ComplexMultiplyByVectorKernel << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}

	void d_ComplexMultiplyByVector(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, NextMultipleOf(elements, 32));
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768), batch);
		ComplexMultiplyByVectorKernel << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}

	void d_ComplexDivideByVector(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768), batch);
		ComplexDivideByVectorKernel << <grid, (uint)TpB >> > (d_input, d_divisors, d_output, elements);
	}

	void d_ComplexDivideSafeByVector(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768), batch);
		ComplexDivideSafeByVectorKernel << <grid, (uint)TpB >> > (d_input, d_divisors, d_output, elements);
	}

	void d_ComplexMultiplyByConjVector(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, NextMultipleOf(elements, 32));
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768), batch);
		ComplexMultiplyByConjVectorKernel << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}

	void d_ComplexMultiplyByScalar(tcomplex* d_input, tcomplex* d_output, size_t elements, tfloat multiplicator)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		ComplexMultiplyByScalarKernel << <grid, (uint)TpB >> > (d_input, d_output, elements, multiplicator);
	}

	void d_ComplexMultiplyByScalar(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		ComplexMultiplyByScalarKernel << <grid, (uint)TpB >> > (d_input, d_output, elements, multiplicator);
	}

	void d_ComplexMultiplyByConjScalar(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		ComplexMultiplyByConjScalarKernel << <grid, (uint)TpB >> > (d_input, d_output, elements, cconj(multiplicator));
	}

	void d_ComplexMultiplyByScalar(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		ComplexMultiplyByScalarKernel << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}

	void d_ComplexMultiplyByScalar(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		ComplexMultiplyByScalarKernel << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}

	void d_ComplexMultiplyByConjScalar(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		ComplexMultiplyByConjScalarKernel << <grid, (uint)TpB >> > (d_input, d_multiplicators, d_output, elements);
	}

	__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements)
	{
		tfloat val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_multiplicators[id];
			d_output[id + elements * blockIdx.y].x = d_input[id + elements * blockIdx.y].x * val;
			d_output[id + elements * blockIdx.y].y = d_input[id + elements * blockIdx.y].y * val;
		}
	}

	__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements)
	{
		tcomplex val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_multiplicators[id];
			d_output[id + elements * blockIdx.y] = cmul(d_input[id + elements * blockIdx.y], val);
		}
	}

	__global__ void ComplexDivideByVectorKernel(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements)
	{
		d_input += elements * blockIdx.y;
		d_output += elements * blockIdx.y;

		tfloat val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = (tfloat)1 / d_divisors[id];
			d_output[id] = d_input[id] * val;
		}
	}

	__global__ void ComplexDivideSafeByVectorKernel(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements)
	{
		d_input += elements * blockIdx.y;
		d_output += elements * blockIdx.y;

		tfloat val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_divisors[id];
			if (abs(val) < 1e-15)
				val = 0;
			else
				val = (tfloat)1 / val;
			d_output[id] = d_input[id] * val;
		}
	}

	__global__ void ComplexMultiplyByConjVectorKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements)
	{
		tcomplex val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = cconj(d_multiplicators[id]);
			d_output[id + elements * blockIdx.y] = cmul(d_input[id + elements * blockIdx.y], val);
		}
	}

	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tfloat multiplicator)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			d_output[id].x = d_input[id].x * multiplicator;
			d_output[id].y = d_input[id].y * multiplicator;
		}
	}

	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = cmul(d_input[id], multiplicator);
	}

	__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = cmul(d_input[id], multiplicator);
	}

	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements)
	{
		__shared__ tfloat scalar;
		if (threadIdx.x == 0)
			scalar = d_multiplicators[blockIdx.y];
		__syncthreads();

		size_t offset = elements * blockIdx.y;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			d_output[id + offset].x = d_input[id + offset].x * scalar;
			d_output[id + offset].y = d_input[id + offset].y * scalar;
		}
	}

	__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements)
	{
		__shared__ tcomplex scalar;
		if (threadIdx.x == 0)
			scalar = d_multiplicators[blockIdx.y];
		__syncthreads();

		size_t offset = elements * blockIdx.y;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id + offset] = cmul(d_input[id + offset], scalar);
	}

	__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements)
	{
		tcomplex scalar;
		scalar = cconj(d_multiplicators[blockIdx.y]);

		size_t offset = elements * blockIdx.y;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id + offset] = cmul(d_input[id + offset], scalar);
	}


	////////////
	//Division//
	////////////

	template <class T> void d_DivideByVector(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		DivideByVectorKernel<T> << <grid, (uint)TpB >> > (d_input, d_divisors, d_output, elements, batch);
	}
	template void d_DivideByVector<float>(float* d_input, float* d_divisors, float* d_output, size_t elements, int batch);
	template void d_DivideByVector<double>(double* d_input, double* d_divisors, double* d_output, size_t elements, int batch);
	template void d_DivideByVector<int>(int* d_input, int* d_divisors, int* d_output, size_t elements, int batch);

	template <class T> void d_DivideSafeByVector(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		DivideSafeByVectorKernel<T> << <grid, (uint)TpB >> > (d_input, d_divisors, d_output, elements, batch);
	}
	template void d_DivideSafeByVector<float>(float* d_input, float* d_divisors, float* d_output, size_t elements, int batch);
	template void d_DivideSafeByVector<double>(double* d_input, double* d_divisors, double* d_output, size_t elements, int batch);
	template void d_DivideSafeByVector<int>(int* d_input, int* d_divisors, int* d_output, size_t elements, int batch);

	template <class T> void d_DivideByScalar(T* d_input, T* d_output, size_t elements, T divisor)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		DivideByScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements, divisor);
	}
	template void d_DivideByScalar<float>(float* d_input, float* d_output, size_t elements, float divisor);
	template void d_DivideByScalar<double>(double* d_input, double* d_output, size_t elements, double divisor);
	template void d_DivideByScalar<int>(int* d_input, int* d_output, size_t elements, int divisor);

	template <class T> void d_DivideByScalar(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		DivideByScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_divisors, d_output, elements);
	}
	template void d_DivideByScalar<float>(float* d_input, float* d_divisors, float* d_output, size_t elements, int batch);
	template void d_DivideByScalar<double>(double* d_input, double* d_divisors, double* d_output, size_t elements, int batch);
	template void d_DivideByScalar<int>(int* d_input, int* d_divisors, int* d_output, size_t elements, int batch);

	template <class T> __global__ void DivideByVectorKernel(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch)
	{
		T val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_divisors[id];
			for (size_t n = 0; n < batch; n++)
				d_output[id + elements * n] = d_input[id + elements * n] / val;
		}
	}

	template <class T> __global__ void DivideSafeByVectorKernel(T* d_input, T* d_divisors, T* d_output, size_t elements, int batch)
	{
		T val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_divisors[id];
			if (abs(val) > (T)1e-15)
				for (size_t n = 0; n < batch; n++)
					d_output[id + elements * n] = d_input[id + elements * n] / val;
			else
				for (size_t n = 0; n < batch; n++)
					d_output[id + elements * n] = (T)0;
		}
	}

	template <class T> __global__ void DivideByScalarKernel(T* d_input, T* d_output, size_t elements, T divisor)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_input[id] / divisor;
	}

	template <class T> __global__ void DivideByScalarKernel(T* d_input, T* d_divisors, T* d_output, size_t elements)
	{
		__shared__ T scalar;
		if (threadIdx.x == 0)
			scalar = d_divisors[blockIdx.y];
		__syncthreads();

		size_t offset = elements * blockIdx.y;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id + offset] = d_input[id + offset] / scalar;
	}

	////////////
	//Addition//
	////////////

	template <class T> void d_AddVector(T* d_input, T* d_summands, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		AddVectorKernel<T> << <grid, (uint)TpB >> > (d_input, d_summands, d_output, elements, batch);
	}
	template void d_AddVector<half>(half* d_input, half* d_summands, half* d_output, size_t elements, int batch);
	template void d_AddVector<float>(float* d_input, float* d_summands, float* d_output, size_t elements, int batch);
	template void d_AddVector<double>(double* d_input, double* d_summands, double* d_output, size_t elements, int batch);
	template void d_AddVector<int>(int* d_input, int* d_summands, int* d_output, size_t elements, int batch);

	template <class T> void d_AddScalar(T* d_input, T* d_output, size_t elements, T summand)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		AddScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements, summand);
	}
	template void d_AddScalar<half>(half* d_input, half* d_output, size_t elements, half summand);
	template void d_AddScalar<float>(float* d_input, float* d_output, size_t elements, float summand);
	template void d_AddScalar<double>(double* d_input, double* d_output, size_t elements, double summand);
	template void d_AddScalar<int>(int* d_input, int* d_output, size_t elements, int summand);

	template <class T> void d_AddScalar(T* d_input, T* d_summands, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		AddScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_summands, d_output, elements);
	}
	template void d_AddScalar<half>(half* d_input, half* d_summands, half* d_output, size_t elements, int batch);
	template void d_AddScalar<float>(float* d_input, float* d_summands, float* d_output, size_t elements, int batch);
	template void d_AddScalar<double>(double* d_input, double* d_summands, double* d_output, size_t elements, int batch);
	template void d_AddScalar<int>(int* d_input, int* d_summands, int* d_output, size_t elements, int batch);

	template <class T> __global__ void AddVectorKernel(T* d_input, T* d_summands, T* d_output, size_t elements, int batch)
	{
		T val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_summands[id];
			for (size_t n = 0; n < batch; n++)
				d_output[id + elements * n] = d_input[id + elements * n] + val;
		}
	}

	template<> __global__ void AddVectorKernel<half>(half* d_input, half* d_summands, half* d_output, size_t elements, int batch)
	{
		float val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = __half2float(d_summands[id]);
			for (size_t n = 0; n < batch; n++)
				d_output[id + elements * n] = __float2half(__half2float(d_input[id + elements * n]) + val);
		}
	}

	template <class T> __global__ void AddScalarKernel(T* d_input, T* d_output, size_t elements, T summand)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_input[id] + summand;
	}

	template<> __global__ void AddScalarKernel<half>(half* d_input, half* d_output, size_t elements, half summand)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = __float2half(__half2float(d_input[id]) + __half2float(summand));
	}

	template <class T> __global__ void AddScalarKernel(T* d_input, T* d_summands, T* d_output, size_t elements)
	{
		T scalar = d_summands[blockIdx.y];

		d_input += blockIdx.y * elements;
		d_output += blockIdx.y * elements;

		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_input[id] + scalar;
	}

	template<> __global__ void AddScalarKernel<half>(half* d_input, half* d_summands, half* d_output, size_t elements)
	{
		float scalar = __half2float(d_summands[blockIdx.y]);

		d_input += blockIdx.y * elements;
		d_output += blockIdx.y * elements;

		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = __float2half(__half2float(d_input[id]) + scalar);
	}


	///////////////
	//Subtraction//
	///////////////

	template <class T> void d_SubtractVector(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		SubtractVectorKernel<T> << <grid, (uint)TpB >> > (d_input, d_subtrahends, d_output, elements, batch);
	}
	template void d_SubtractVector<half>(half* d_input, half* d_subtrahends, half* d_output, size_t elements, int batch);
	template void d_SubtractVector<float>(float* d_input, float* d_subtrahends, float* d_output, size_t elements, int batch);
	template void d_SubtractVector<double>(double* d_input, double* d_subtrahends, double* d_output, size_t elements, int batch);
	template void d_SubtractVector<int>(int* d_input, int* d_subtrahends, int* d_output, size_t elements, int batch);

	template <class T> void d_SubtractScalar(T* d_input, T* d_output, size_t elements, T subtrahend)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		SubtractScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements, subtrahend);
	}
	template void d_SubtractScalar<half>(half* d_input, half* d_output, size_t elements, half subtrahend);
	template void d_SubtractScalar<float>(float* d_input, float* d_output, size_t elements, float subtrahend);
	template void d_SubtractScalar<double>(double* d_input, double* d_output, size_t elements, double subtrahend);
	template void d_SubtractScalar<int>(int* d_input, int* d_output, size_t elements, int subtrahend);

	template <class T> void d_SubtractScalar(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks, batch);
		SubtractScalarKernel<T> << <grid, (uint)TpB >> > (d_input, d_subtrahends, d_output, elements);
	}
	template void d_SubtractScalar<half>(half* d_input, half* d_subtrahends, half* d_output, size_t elements, int batch);
	template void d_SubtractScalar<float>(float* d_input, float* d_subtrahends, float* d_output, size_t elements, int batch);
	template void d_SubtractScalar<double>(double* d_input, double* d_subtrahends, double* d_output, size_t elements, int batch);
	template void d_SubtractScalar<int>(int* d_input, int* d_subtrahends, int* d_output, size_t elements, int batch);

	template <class T> __global__ void SubtractVectorKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch)
	{
		T val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = d_subtrahends[id];
			for (size_t n = 0; n < batch; n++)
				d_output[id + elements * n] = d_input[id + elements * n] - val;
		}
	}

	template<> __global__ void SubtractVectorKernel<half>(half* d_input, half* d_subtrahends, half* d_output, size_t elements, int batch)
	{
		float val;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			val = __half2float(d_subtrahends[id]);
			for (size_t n = 0; n < batch; n++)
				d_output[id + elements * n] = __float2half(__half2float(d_input[id + elements * n]) - val);
		}
	}

	template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_output, size_t elements, T subtrahend)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_input[id] - subtrahend;
	}

	template<> __global__ void SubtractScalarKernel<half>(half* d_input, half* d_output, size_t elements, half subtrahend)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = __float2half(__half2float(d_input[id]) - __half2float(subtrahend));
	}

	template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements)
	{
		T scalar = d_subtrahends[blockIdx.y];

		d_input += blockIdx.y * elements;
		d_output += blockIdx.y * elements;

		size_t gridsize = blockDim.x * gridDim.x;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += gridsize)
			d_output[id] = d_input[id] - scalar;
	}

	template<> __global__ void SubtractScalarKernel<half>(half* d_input, half* d_subtrahends, half* d_output, size_t elements)
	{
		float scalar = __half2float(d_subtrahends[blockIdx.y]);

		d_input += blockIdx.y * elements;
		d_output += blockIdx.y * elements;

		size_t gridsize = blockDim.x * gridDim.x;
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += gridsize)
			d_output[id] = __float2half(__half2float(d_input[id]) - scalar);
	}


	//////////
	//Square//
	//////////

	template <class T> void d_Square(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)32768);
		dim3 grid = dim3((uint)totalblocks);
		SquareKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Square<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements);
	template void d_Square<int>(int* d_input, int* d_output, size_t elements);

	template <class T> __global__ void SquareKernel(T* d_input, T* d_output, size_t elements)
	{
		T val;
		int gridsize = blockDim.x * gridDim.x;
		for (int id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += gridsize)
		{
			val = d_input[id];
			d_output[id] = val * val;
		}
	}


	///////////////
	//Square root//
	///////////////

	template <class T> void d_Sqrt(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		SqrtKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Sqrt<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements);
	//template void d_Sqrt<int>(int* d_input, int* d_output, size_t elements);

	template <class T> __global__ void SqrtKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = sqrt(d_input[id]);
	}


	/////////
	//Power//
	/////////

	template <class T> void d_Pow(T* d_input, T* d_output, size_t elements, T exponent)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		PowKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements, exponent);
	}
	template void d_Pow<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat exponent);

	template <class T> __global__ void PowKernel(T* d_input, T* d_output, size_t elements, T exponent)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = pow(d_input[id], exponent);
	}


	///////
	//Abs//
	///////

	template <class T> void d_Abs(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		AbsKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Abs<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements);

	void d_Abs(tcomplex* d_input, tfloat* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		AbsKernel << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}

	template <class T> __global__ void AbsKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = abs(d_input[id]);
	}

	__global__ void AbsKernel(tcomplex* d_input, tfloat* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			tfloat re = d_input[id].x;
			tfloat im = d_input[id].y;
			d_output[id] = sqrt(re * re + im * im);
		}
	}


	///////////
	//Inverse//
	///////////

	template <class T> void d_Inv(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		InvKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Inv<float>(float* d_input, float* d_output, size_t elements);
	template void d_Inv<double>(double* d_input, double* d_output, size_t elements);

	template <class T> __global__ void InvKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			if (d_input[id] != (T)0)
				d_output[id] = (T)1 / d_input[id];
	}


	/////////////
	//Logarithm//
	/////////////

	template <class T> void d_Log(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		LogKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Log<float>(float* d_input, float* d_output, size_t elements);
	template void d_Log<double>(double* d_input, double* d_output, size_t elements);

	template <class T> __global__ void LogKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = log(d_input[id]);
	}


	////////////
	//Exponent//
	////////////

	template <class T> void d_Exp(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		ExpKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Exp<float>(float* d_input, float* d_output, size_t elements);
	template void d_Exp<double>(double* d_input, double* d_output, size_t elements);

	template <class T> __global__ void ExpKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = exp(d_input[id]);
	}


	/////////
	//1 - x//
	/////////

	template <class T> void d_OneMinus(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		OneMinusKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_OneMinus<float>(float* d_input, float* d_output, size_t elements);
	template void d_OneMinus<double>(double* d_input, double* d_output, size_t elements);
	template void d_OneMinus<int>(int* d_input, int* d_output, size_t elements);
	template void d_OneMinus<short>(short* d_input, short* d_output, size_t elements);
	template void d_OneMinus<char>(char* d_input, char* d_output, size_t elements);

	template <class T> __global__ void OneMinusKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = (T)1 - d_input[id];
	}


	///////////
	//sign(x)//
	///////////

	template <class T> void d_Sign(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		SignKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Sign<float>(float* d_input, float* d_output, size_t elements);
	template void d_Sign<double>(double* d_input, double* d_output, size_t elements);
	template void d_Sign<int>(int* d_input, int* d_output, size_t elements);
	template void d_Sign<short>(short* d_input, short* d_output, size_t elements);
	template void d_Sign<char>(char* d_input, char* d_output, size_t elements);

	template <class T> __global__ void SignKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = sgn(d_input[id]);
	}


	////////////////
	//Trigonometry//
	////////////////

	template <class T> void d_Cos(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		CosKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Cos<float>(float* d_input, float* d_output, size_t elements);
	template void d_Cos<double>(double* d_input, double* d_output, size_t elements);

	template <class T> __global__ void CosKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = cos(d_input[id]);
	}

	template <class T> void d_Sin(T* d_input, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		SinKernel<T> << <grid, (uint)TpB >> > (d_input, d_output, elements);
	}
	template void d_Sin<float>(float* d_input, float* d_output, size_t elements);
	template void d_Sin<double>(double* d_input, double* d_output, size_t elements);

	template <class T> __global__ void SinKernel(T* d_input, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = sin(d_input[id]);
	}


	//////////////////////
	//Fused multiply-add//
	//////////////////////
	
	template <class T> void d_MultiplyAdd(T* d_mult1, T* d_mult2, T* d_summand, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		MultiplyAddKernel<T> << <grid, (uint)TpB >> > (d_mult1, d_mult2, d_summand, d_output, elements);
	}
	template void d_MultiplyAdd<float>(float* d_mult1, float* d_mult2, float* d_summand, float* d_output, size_t elements);
	template void d_MultiplyAdd<double>(double* d_mult1, double* d_mult2, double* d_summand, double* d_output, size_t elements);
	template void d_MultiplyAdd<int>(int* d_mult1, int* d_mult2, int* d_summand, int* d_output, size_t elements);
	template void d_MultiplyAdd<short>(short* d_mult1, short* d_mult2, short* d_summand, short* d_output, size_t elements);
	template void d_MultiplyAdd<char>(char* d_mult1, char* d_mult2, char* d_summand, char* d_output, size_t elements);

	template <class T> __global__ void MultiplyAddKernel(T* d_mult1, T* d_mult2, T* d_summand, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = d_mult1[id] * d_mult2[id] + d_summand[id];
	}


	/////////////////////////////////
	//Complex number representation//
	/////////////////////////////////

	void d_ComplexPolarToCart(tcomplex* d_polar, tcomplex* d_cart, size_t elements)
	{
		int TpB = tmin((size_t)256, elements);
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)8192));
		ComplexPolarToCartKernel << <grid, TpB >> > (d_polar, d_cart, elements);
	}

	__global__ void ComplexPolarToCartKernel(tcomplex* d_polar, tcomplex* d_cart, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
#ifndef GTOM_DOUBLE
			d_cart[id] = make_hipComplex(cos(d_polar[id].y) * d_polar[id].x, sin(d_polar[id].y) * d_polar[id].x);
#else
			d_cart[id] = make_hipDoubleComplex(cos(d_polar[id].y) * d_polar[id].x, sin(d_polar[id].y) * d_polar[id].x);
#endif
	}

	void d_ComplexCartToPolar(tcomplex* d_cart, tcomplex* d_polar, size_t elements)
	{
		int TpB = tmin((size_t)256, elements);
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)8192));
		ComplexCartToPolarKernel << <grid, TpB >> > (d_cart, d_polar, elements);
	}

	__global__ void ComplexCartToPolarKernel(tcomplex* d_cart, tcomplex* d_polar, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
#ifndef GTOM_DOUBLE
			d_cart[id] = make_hipComplex(sqrt(d_cart[id].x * d_cart[id].x + d_cart[id].y * d_cart[id].y), atan2(d_cart[id].y, d_cart[id].x));
#else
			d_cart[id] = make_hipDoubleComplex(sqrt(d_cart[id].x * d_cart[id].x + d_cart[id].y * d_cart[id].y), atan2(d_cart[id].y, d_cart[id].x));
#endif
	}

	void d_ComplexNormalize(tcomplex* d_input, tcomplex* d_output, size_t elements)
	{
		int TpB = tmin((size_t)256, elements);
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)8192));
		ComplexNormalizeKernel << <grid, TpB >> > (d_input, d_output, elements);
	}

	__global__ void ComplexNormalizeKernel(tcomplex* d_input, tcomplex* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
		{
			tcomplex input = d_input[id];

#ifndef GTOM_DOUBLE
			tfloat magnitude = hypotf(input.x, input.y);
			if (magnitude > 0.0f)
				magnitude = 1.0f / magnitude;
			d_output[id] = make_hipComplex(input.x * magnitude, input.y * magnitude);
#else
			tfloat magnitude = hypot(input.x, input.y);
			if (magnitude > 0.0)
				magnitude = 1.0 / magnitude;
			d_cart[id] = make_hipDoubleComplex(input.x * magnitude, input.y * magnitude);
#endif
		}
	}


	///////////////
	//Min/Max ops//
	///////////////

	template <class T> void d_MaxOp(T* d_input1, T* d_input2, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		MaxOpKernel<T> << <grid, (uint)TpB >> > (d_input1, d_input2, d_output, elements);
	}
	template void d_MaxOp<int>(int* d_input1, int* d_input2, int* d_output, size_t elements);
	template void d_MaxOp<float>(float* d_input1, float* d_input2, float* d_output, size_t elements);
	template void d_MaxOp<double>(double* d_input1, double* d_input2, double* d_output, size_t elements);

	template <class T> void d_MaxOp(T* d_input1, T input2, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		MaxOpKernel<T> << <grid, (uint)TpB >> > (d_input1, input2, d_output, elements);
	}
	template void d_MaxOp<int>(int* d_input1, int input2, int* d_output, size_t elements);
	template void d_MaxOp<float>(float* d_input1, float input2, float* d_output, size_t elements);
	template void d_MaxOp<double>(double* d_input1, double input2, double* d_output, size_t elements);

	template <class T> void d_MinOp(T* d_input1, T* d_input2, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		MinOpKernel<T> << <grid, (uint)TpB >> > (d_input1, d_input2, d_output, elements);
	}
	template void d_MinOp<int>(int* d_input1, int* d_input2, int* d_output, size_t elements);
	template void d_MinOp<float>(float* d_input1, float* d_input2, float* d_output, size_t elements);
	template void d_MinOp<double>(double* d_input1, double* d_input2, double* d_output, size_t elements);

	template <class T> void d_MinOp(T* d_input1, T input2, T* d_output, size_t elements)
	{
		size_t TpB = tmin((size_t)256, elements);
		size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
		dim3 grid = dim3((uint)totalblocks);
		MinOpKernel<T> << <grid, (uint)TpB >> > (d_input1, input2, d_output, elements);
	}
	template void d_MinOp<int>(int* d_input1, int input2, int* d_output, size_t elements);
	template void d_MinOp<float>(float* d_input1, float input2, float* d_output, size_t elements);
	template void d_MinOp<double>(double* d_input1, double input2, double* d_output, size_t elements);

	template <class T> __global__ void MaxOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = max(d_input1[id], d_input2[id]);
	}

	template <class T> __global__ void MaxOpKernel(T* d_input1, T input2, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = max(d_input1[id], input2);
	}

	template <class T> __global__ void MinOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = min(d_input1[id], d_input2[id]);
	}

	template <class T> __global__ void MinOpKernel(T* d_input1, T input2, T* d_output, size_t elements)
	{
		for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
			id < elements;
			id += blockDim.x * gridDim.x)
			d_output[id] = min(d_input1[id], input2);
	}


	////////
	//Misc//
	////////

	size_t NextPow2(size_t x)
	{
		--x;
		x |= x >> 1;
		x |= x >> 2;
		x |= x >> 4;
		x |= x >> 8;
		x |= x >> 16;
		return ++x;
	}

	bool IsPow2(size_t x)
	{
		return x && !(x & (x - 1));
	}
}