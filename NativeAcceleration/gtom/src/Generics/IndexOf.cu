#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	__global__ void FirstIndexOfLinearKernel(tfloat* d_input, tfloat* d_output, size_t elements, tfloat value);
	__global__ void HalfBitFSCKernel(tfloat* d_input, tfloat* d_output, size_t elements);
	__global__ void FirstMinimumLinearKernel(tfloat* d_input, tfloat* d_output, size_t elements);

	template<class T> __global__ void BiggerThanKernel(tfloat* d_input, T* d_output, size_t elements, tfloat value);
	template<class T> __global__ void SmallerThanKernel(tfloat* d_input, T* d_output, size_t elements, tfloat value);
	template<class T> __global__ void IsBetweenKernel(tfloat* d_input, T* d_output, size_t elements, tfloat minval, tfloat maxval);


	//////////////////
	//First Index Of//
	//////////////////

	void d_FirstIndexOf(tfloat* d_input, tfloat* d_output, size_t elements, tfloat value, T_INTERP_MODE mode, int batch)
	{
		if (mode == T_INTERP_LINEAR)
		{
			int TpB = tmin(NextMultipleOf(elements, 32), (size_t)256);
			dim3 grid = dim3(batch);
			FirstIndexOfLinearKernel << <grid, TpB >> > (d_input, d_output, elements, value);
		}
		else
			throw;
	}

	__global__ void FirstIndexOfLinearKernel(tfloat* d_input, tfloat* d_output, size_t elements, tfloat value)
	{
		d_input += blockIdx.x * elements;

		__shared__ tfloat indices[256];
		__shared__ bool found, anybigger, nan;
		if (threadIdx.x == 0)
		{
			found = false;
			anybigger = false;
			nan = false;
		}
		__syncthreads();

		tfloat index = (tfloat)(elements + 1);
		tfloat current, next;
		for (size_t n = threadIdx.x; n < elements - 1; n += blockDim.x)
		{
			if (found)
				break;

			current = d_input[n];
			next = d_input[n + 1];
			if (isnan(current) || isnan(next))
			{
				nan = true;
				continue;
			}

			if (value < current)
				anybigger = true;

			if ((value <= current && value >= next) || (value >= current && value <= next))
			{
				index = (tfloat)n + max(min((value - current) / (next - current + (tfloat)0.00001), 1.0f), 0.0f);
				found = true;
				break;
			}
		}

		indices[threadIdx.x] = index;

		__syncthreads();

		if (threadIdx.x == 0)
		{
			for (int t = 1; t < min((long long)elements, (long long)blockDim.x); t++)
				index = min(index, indices[t]);

			if (found)
				d_output[blockIdx.x] = max(index, 1.0f);
			else if (anybigger)
				d_output[blockIdx.x] = (tfloat)elements;
			else if (nan)
				d_output[blockIdx.x] = (tfloat)1;
			else
				d_output[blockIdx.x] = (tfloat)1;
		}
	}

	////////////////////////////////////////////////////////////////////
	//FSC intersection according to half-bit criterion (van Heel 2005)//
	////////////////////////////////////////////////////////////////////

	void d_IntersectHalfBitFSC(tfloat* d_input, tfloat* d_output, size_t elements, tfloat* d_structurefraction, int batch)
	{
		int TpB = tmin(NextMultipleOf(elements, 32), (size_t)256);
		dim3 grid = dim3(batch);
		HalfBitFSCKernel << <grid, TpB >> > (d_input, d_output, elements);
	}

	__global__ void HalfBitFSCKernel(tfloat* d_input, tfloat* d_output, size_t elements)
	{
		d_input += blockIdx.x * elements;

		__shared__ tfloat indices[256];
		__shared__ bool found, anybigger, nan;
		if (threadIdx.x == 0)
		{
			found = false;
			anybigger = false;
			nan = false;
		}
		__syncthreads();

		tfloat index = (tfloat)(elements + 1);
		tfloat current, next;
		for (size_t n = threadIdx.x; n < elements - 1; n += blockDim.x)
		{
			if (found)
				break;

			current = d_input[n];
			next = d_input[n + 1];
			if (isnan(current) || isnan(next))
			{
				nan = true;
				continue;
			}

			tfloat rootn0 = sqrt((tfloat)n);
			tfloat rootn1 = sqrt((tfloat)(n + 1));
			float value0 = n == 0 ? 0.99f : min(0.99f, (0.2071f + 1.9102f / rootn0) / (1.2071f + 0.9102f / rootn0));
			float value1 = min(0.99f, (0.2071f + 1.9102f / rootn1) / (1.2071f + 0.9102f / rootn1));

			if (value0 < current && value1 < next)
				anybigger = true;

			if ((value0 <= current && value1 >= next) || (value0 >= current && value1 <= next))
			{
				index = (tfloat)n;
				tfloat denominator = (value1 - value0) - (next - current);
				if (abs(denominator) > 0.00001f)	//not parallel
					index = (tfloat)n + (current - value0) / denominator;
				found = true;
				break;
			}
		}

		indices[threadIdx.x] = index;

		__syncthreads();

		if (threadIdx.x == 0)
		{
			for (int t = 1; t < min((long long)elements, (long long)blockDim.x); t++)
				index = min(index, indices[t]);

			if (found)
				d_output[blockIdx.x] = max(index, 0.0f);
			else if (anybigger)
				d_output[blockIdx.x] = (tfloat)elements;
			else if (nan)
				d_output[blockIdx.x] = (tfloat)-1;
			else
				d_output[blockIdx.x] = (tfloat)0;
		}
	}

	//////////////////////////
	//Index of first minimum//
	//////////////////////////

	void d_FirstMinimum(tfloat* d_input, tfloat* d_output, size_t elements, T_INTERP_MODE mode, int batch)
	{
		if (mode == T_INTERP_LINEAR)
		{
			int TpB = tmin(NextMultipleOf(elements, 32), (size_t)256);
			dim3 grid = dim3(batch);
			FirstMinimumLinearKernel << <grid, TpB >> > (d_input, d_output, elements);
		}
		else
			throw;
	}

	__global__ void FirstMinimumLinearKernel(tfloat* d_input, tfloat* d_output, size_t elements)
	{
		d_input += blockIdx.x * elements;

		__shared__ tfloat indices[256];
		__shared__ bool found;
		if (threadIdx.x == 0)
		{
			found = false;
		}
		__syncthreads();

		tfloat index = (tfloat)(elements + 1);
		tfloat left, right, current;
		for (size_t n = threadIdx.x + 1; n < elements - 1; n += blockDim.x)
		{
			left = d_input[n - 1];
			current = d_input[n];
			right = d_input[n + 1];

			if (isnan(left) || isnan(current) || isnan(right))
			{
				continue;
			}

			if (current < left && current < right)
			{
				index = (tfloat)n;
				found = true;
				break;
			}
		}

		indices[threadIdx.x] = index;

		__syncthreads();

		if (threadIdx.x == 0)
		{
			for (int t = 1; t < min((long long)elements, (long long)blockDim.x); t++)
				index = min(index, indices[t]);

			if (found)
				d_output[blockIdx.x] = max(index, 0.0f);
			else
				d_output[blockIdx.x] = (tfloat)-1;
		}
	}


	//////////////////
	//Is Bigger Than//
	//////////////////

	template<class T> void d_BiggerThan(tfloat* d_input, T* d_output, size_t elements, tfloat value)
	{
		int TpB = tmin((size_t)256, NextMultipleOf(elements, 32));
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768));
		BiggerThanKernel << <grid, TpB >> > (d_input, d_output, elements, value);
	}
	template void d_BiggerThan<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat value);
	template void d_BiggerThan<char>(tfloat* d_input, char* d_output, size_t elements, tfloat value);

	template<class T> __global__ void BiggerThanKernel(tfloat* d_input, T* d_output, size_t elements, tfloat value)
	{
		for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			d_output[i] = d_input[i] > value ? (T)1 : (T)0;
	}


	///////////////////
	//Is Smaller Than//
	///////////////////

	template<class T> void d_SmallerThan(tfloat* d_input, T* d_output, size_t elements, tfloat value)
	{
		int TpB = tmin((size_t)256, NextMultipleOf(elements, 32));
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768));
		SmallerThanKernel << <grid, TpB >> > (d_input, d_output, elements, value);
	}
	template void d_SmallerThan<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat value);
	template void d_SmallerThan<char>(tfloat* d_input, char* d_output, size_t elements, tfloat value);

	template<class T> __global__ void SmallerThanKernel(tfloat* d_input, T* d_output, size_t elements, tfloat value)
	{
		for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			d_output[i] = d_input[i] < value ? (T)1 : (T)0;
	}


	//////////////
	//Is Between//
	//////////////

	template<class T> void d_IsBetween(tfloat* d_input, T* d_output, size_t elements, tfloat minval, tfloat maxval)
	{
		int TpB = tmin((size_t)256, NextMultipleOf(elements, 32));
		dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768));
		IsBetweenKernel << <grid, TpB >> > (d_input, d_output, elements, minval, maxval);
	}
	template void d_IsBetween<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat minval, tfloat maxval);
	template void d_IsBetween<char>(tfloat* d_input, char* d_output, size_t elements, tfloat minval, tfloat maxval);

	template<class T> __global__ void IsBetweenKernel(tfloat* d_input, T* d_output, size_t elements, tfloat minval, tfloat maxval)
	{
		for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			d_output[i] = (d_input[i] < maxval && d_input[i] >= minval) ? (T)1 : (T)0;
	}
}