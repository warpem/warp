#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/FFT.cuh"
#include "gtom/include/Generics.cuh"
#include "gtom/include/Helper.cuh"
#include "gtom/include/Masking.cuh"
#include "gtom/include/Resolution.cuh"


namespace gtom
{
	///////////////////////////
	//CUDA kernel declaration//
	///////////////////////////

	__global__ void LocalFilterKernel(tcomplex* d_input,
										tcomplex* d_output,
										uint sidelength,
										uint sidelengthft,
										tfloat angpix,
										tfloat* d_resolution,
										tfloat* d_filterramps,
										int rampsoversample,
										tfloat* d_debugfsc);

	///////////////////////////////////
	//Local Fourier Shell Correlation//
	///////////////////////////////////

	void d_LocalFilter(tfloat* d_input,
						tfloat* d_filtered,
						int3 dimsvolume,
						tfloat* d_resolution,
						int windowsize,
						tfloat angpix,
						tfloat* d_filterramps,
						int rampsoversample)
	{
		// dimsvolume sans the region where window around position of interest would exceed the volume
		int3 dimsaccessiblevolume = toInt3(dimsvolume.x - windowsize, dimsvolume.y - windowsize, dimsvolume.z - windowsize);
		int3 dimswindow = toInt3(windowsize, windowsize, windowsize);

		uint batchmemory = 512 << 20;
		uint windowmemory = Elements(dimswindow) * sizeof(tfloat);
		uint batchsize = batchmemory / windowmemory;

		tfloat* d_accessibleresolution = CudaMallocValueFilled(Elements(dimsaccessiblevolume), (tfloat)0);
		tfloat* d_accessiblecorrected = CudaMallocValueFilled(Elements(dimsaccessiblevolume), (tfloat)0);

		d_Pad(d_resolution, d_accessibleresolution, dimsvolume, dimsaccessiblevolume, T_PAD_VALUE, (tfloat)0);

		// Allocate buffers for batch window extraction
		tfloat *d_extracts1;
		hipMalloc((void**)&d_extracts1, Elements(dimswindow) * batchsize * sizeof(tfloat));

		// ... and their FT
		tcomplex* d_extractsft1;
		hipMalloc((void**)&d_extractsft1, ElementsFFT(dimswindow) * batchsize * sizeof(tcomplex));

		// Hann mask for extracted portions
		tfloat* d_mask = CudaMallocValueFilled(Elements(dimswindow), (tfloat)1);
		d_HannMask(d_mask, d_mask, dimswindow, NULL, NULL);
		//d_WriteMRC(d_mask, dimswindow, "d_mask.mrc");

		// Positions at which the windows will be extracted
		int3* h_extractorigins;
		hipHostMalloc((void**)&h_extractorigins, batchsize * sizeof(int3));
		int3* d_extractorigins;
		hipMalloc((void**)&d_extractorigins, batchsize * sizeof(int3));

		// Batch FFT for extracted windows
		hipfftHandle planforw = d_FFTR2CGetPlan(3, dimswindow, batchsize);
		hipfftHandle planback = d_IFFTC2RGetPlan(3, dimswindow, batchsize);

		int elementsvol = Elements(dimsaccessiblevolume);
		int elementsslice = dimsaccessiblevolume.x * dimsaccessiblevolume.y;
		int elementswindow = Elements(dimswindow);

		d_Inv(d_accessibleresolution, d_accessibleresolution, Elements(dimsaccessiblevolume));
		d_MultiplyByScalar(d_accessibleresolution, d_accessibleresolution, Elements(dimsaccessiblevolume), windowsize * angpix);

		for (int i = 0; i < elementsvol; i += batchsize)
		{
			uint curbatch = tmin(batchsize, elementsvol - i);

			for (int b = 0; b < curbatch; b++)
			{
				// Set origins for window extraction
				int z = (i + b) / elementsslice;
				int y = ((i + b) % elementsslice) / dimsaccessiblevolume.x;
				int x = (i + b) % dimsaccessiblevolume.x;
				h_extractorigins[b] = toInt3(x, y, z);
			}
			hipMemcpy(d_extractorigins, h_extractorigins, curbatch * sizeof(int3), hipMemcpyHostToDevice);

			// Extract windows
			d_ExtractMany(d_input, d_extracts1, dimsvolume, dimswindow, d_extractorigins, false, curbatch);

			// Multiply by Hann mask
			d_MultiplyByVector(d_extracts1, d_mask, d_extracts1, elementswindow, curbatch);

			//d_WriteMRC(d_extracts1, dimswindow, "d_extracts1.mrc");
			//d_WriteMRC(d_extracts2, dimswindow, "d_extracts2.mrc");

			// FFT
			d_FFTR2C(d_extracts1, d_extractsft1, &planforw);

			//tfloat* d_debugfsc = CudaMallocValueFilled(windowsize / 2, (tfloat)0);

			int TpB = 128;
			dim3 grid = dim3(curbatch, 1, 1);
			LocalFilterKernel << <grid, TpB >> > (d_extractsft1,
												d_extractsft1,
												windowsize,
												windowsize / 2 + 1,
												angpix,
												d_accessibleresolution + i,
												d_filterramps,
												rampsoversample,
												NULL);

			// Low-pass and sharpened
			d_IFFTC2R(d_extractsft1, d_extracts1, &planback, dimswindow, curbatch);
			CudaMemcpyStrided(d_accessiblecorrected + i, d_extracts1 + (dimswindow.z / 2 * dimswindow.y + dimswindow.y / 2) * dimswindow.x + dimswindow.x / 2, curbatch, 1, Elements(dimswindow));

			/*tfloat* h_debugfsc = (tfloat*)MallocFromDeviceArray(d_debugfsc, windowsize / 2 * sizeof(tfloat));
			hipFree(d_debugfsc);
			free(h_debugfsc);*/

			//tfloat* h_resolution = (tfloat*)MallocFromDeviceArray(d_accessibleresolution + i, curbatch * sizeof(tfloat));
			//tfloat* h_bfac = (tfloat*)MallocFromDeviceArray(d_accessiblebfactors + i, curbatch * sizeof(tfloat));
			//free(h_resolution);
			//free(h_bfac);

			//progressbar(i, elementsvol, 100);
		}

		d_Pad(d_accessiblecorrected, d_filtered, dimsaccessiblevolume, dimsvolume, T_PAD_VALUE, (tfloat)0);

		hipfftDestroy(planback);
		hipfftDestroy(planforw);


		hipFree(d_accessiblecorrected);
		hipFree(d_accessibleresolution);
		hipFree(d_extractorigins);
		hipFree(d_mask);
		hipFree(d_extractsft1);
		hipFree(d_extracts1);

		hipHostFree(h_extractorigins);
	}

__global__ void LocalFilterKernel(tcomplex* d_input,
									tcomplex* d_output,
									uint sidelength,
									uint sidelengthft,
									tfloat angpix,
									tfloat* d_resolution,
									tfloat* d_filterramps,
									int rampsoversample,
									tfloat* d_debugfsc)
	{
		float cutoffshell;
		cutoffshell = d_resolution[blockIdx.x];

		uint elementsslice = sidelengthft * sidelength;
		uint elementscube = elementsslice * sidelength;

		d_input += elementscube * blockIdx.x;
		d_output += elementscube * blockIdx.x;
		
		// Filter and sharpen the input
		{
			uint sidelengthhalf = sidelength / 2;

			for (uint id = threadIdx.x; id < elementscube; id += 128)
			{
				int idz = (int)(id / elementsslice);
				int idy = (int)((id % elementsslice) / sidelengthft);
				int idx = (int)(id % sidelengthft);

				tfloat rx = idx;
				tfloat ry = idy <= sidelengthhalf ? idy : idy - (int)sidelength;
				tfloat rz = idz <= sidelengthhalf ? idz : idz - (int)sidelength;
				tfloat radius = sqrt(rx * rx + ry * ry + rz * rz);

				int sidehalf = sidelength / 2;

				if (radius >= sidehalf)
				{
					d_output[id] = make_hipComplex(0, 0);
					continue;
				}

				tfloat ramp00 = d_filterramps[tmin((int)(cutoffshell * rampsoversample), sidehalf * rampsoversample - 1) * sidehalf + (int)radius];
				tfloat ramp01 = d_filterramps[tmin((int)(cutoffshell * rampsoversample), sidehalf * rampsoversample - 1) * sidehalf + tmin((int)radius + 1, sidehalf - 1)];

				tfloat ramp10 = d_filterramps[tmin((int)(cutoffshell * rampsoversample) + 1, sidehalf * rampsoversample - 1) * sidehalf + (int)radius];
				tfloat ramp11 = d_filterramps[tmin((int)(cutoffshell * rampsoversample) + 1, sidehalf * rampsoversample - 1) * sidehalf + tmin((int)radius + 1, sidelength / 2 - 1)];

				tfloat ramp0 = lerp(ramp00, ramp01, radius - floor(radius));
				tfloat ramp1 = lerp(ramp10, ramp11, radius - floor(radius));

				tfloat ramp = lerp(ramp0, ramp1, cutoffshell * rampsoversample - floor(cutoffshell * rampsoversample));

				tcomplex val = d_input[id];
				val *= ramp;

				if (isnan(val.x))
					val.x = 0;
				if (isnan(val.y))
					val.y = 0;
				d_output[id] = val;	// sharpened
			}
		}
	}
}