#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Angles.cuh"
#include "gtom/include/Helper.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	template <class T, int ndims> __global__ void IrregularSphereMaskKernel(T* d_input, T* d_output, int3 dims, tfloat sigma, tfloat3 center, cudaTex radiusTex);


	///////////
	//Globals//
	///////////

	// Will replace this global texture reference with texture objects


	////////////////
	//Host methods//
	////////////////

	template <class T> void d_IrregularSphereMask(T* d_input,
		T* d_output,
		int3 dims,
		tfloat* d_radiusmap,
		int2 anglesteps,
		tfloat sigma,
		tfloat3* center,
		int batch)
	{
		// Create pitched memory with proper alignment
		tfloat* d_pitched = NULL;
		int pitchedwidth = anglesteps.x * sizeof(tfloat);
		d_pitched = (tfloat*)CudaMallocAligned2D(anglesteps.x * sizeof(tfloat), anglesteps.y, &pitchedwidth);
		for (int y = 0; y < anglesteps.y; y++)
			hipMemcpy((char*)d_pitched + y * pitchedwidth,
			d_radiusmap + y * anglesteps.x,
			anglesteps.x * sizeof(tfloat),
			hipMemcpyDeviceToDevice);

		// Create a CUDA array and texture object
		hipArray_t radiusArray;
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipMallocArray(&radiusArray, &desc, anglesteps.x, anglesteps.y);
		d_CopyToArray(d_pitched, radiusArray, anglesteps.x, anglesteps.y, 1, sizeof(tfloat), hipMemcpyDeviceToDevice);
		
		// Create texture object with mirror address mode
		cudaTex radiusTex = d_CreateTextureObject(radiusArray, hipFilterModeLinear, hipReadModeElementType, true, hipAddressModeMirror);

		tfloat3 _center = center != NULL ? *center : tfloat3(dims.x / 2, dims.y / 2, dims.z / 2);

		int TpB = min(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, batch);
		if (DimensionCount(dims) <= 2)
			IrregularSphereMaskKernel<T, 2> << <grid, TpB >> > (d_input, d_output, dims, sigma, _center, radiusTex);
		else
			IrregularSphereMaskKernel<T, 3> << <grid, TpB >> > (d_input, d_output, dims, sigma, _center, radiusTex);

		// Clean up
		hipDestroyTextureObject(radiusTex);
		hipFreeArray(radiusArray);
		hipFree(d_pitched);
	}
	template void d_IrregularSphereMask<tfloat>(tfloat* d_input, tfloat* d_output, int3 dims, tfloat* d_radiusmap, int2 anglesteps, tfloat sigma, tfloat3* center, int batch);


	////////////////
	//CUDA kernels//
	////////////////

	template <class T, int ndims> __global__ void IrregularSphereMaskKernel(T* d_input, T* d_output, int3 dims, tfloat sigma, tfloat3 center, cudaTex radiusTex)
	{
		if (threadIdx.x >= dims.x)
			return;

		//For batch mode
		int offset = blockIdx.z * Elements(dims) + blockIdx.y * dims.x * dims.y + blockIdx.x * dims.x;

		int x, y, z;
		float length;
		T maskvalue;

		//Squared y and z distance from center
		y = blockIdx.x - center.y;
		if (ndims > 2)
			z = blockIdx.y - center.z;
		else
			z = 0;

		for (int idx = threadIdx.x; idx < dims.x; idx += blockDim.x)
		{
			x = idx - center.x;

			length = sqrt((float)(x * x + y * y + z * z));

			glm::vec3 direction((float)x / length, (float)y / length, (float)z / length);
			float theta = acos((float)(-direction.x));
			float phi = atan2((float)direction.y / sin(theta), (float)direction.z / sin(theta));

			theta /= PI * 0.5f;
			phi /= PI2;

			// Use modern texture object API
			tfloat radius = tex2D<tfloat>(radiusTex, phi, theta);

			if (length < radius)
				maskvalue = 1;
			else
			{
				//Smooth border
				if (sigma > (tfloat)0)
				{
					maskvalue = exp(-((length - radius) * (length - radius) / (sigma * sigma)));
					if (maskvalue < (tfloat)0.1353)
						maskvalue = 0;
				}
				//Hard border
				else
					maskvalue = max((T)1 - (length - radius), (T)0);
			}

			//Write masked input to output
			d_output[offset + idx] = maskvalue * d_input[offset + idx];
			//d_output[offset + idx] = radius;
		}
	}
}