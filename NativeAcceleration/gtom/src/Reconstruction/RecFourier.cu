#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Angles.cuh"
#include "gtom/include/CTF.cuh"
#include "gtom/include/DeviceFunctions.cuh"
#include "gtom/include/FFT.cuh"
#include "gtom/include/Generics.cuh"
#include "gtom/include/Helper.cuh"
#include "gtom/include/Masking.cuh"
#include "gtom/include/Reconstruction.cuh"
#include "gtom/include/Transformation.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	__global__ void PrecomputeBlobKernel(tfloat* d_precompblob, int paddim, int oripadded, tfloat* d_funcvals, tfloat funcsampling, int funcelements);
	template<class T> __global__ void DecenterKernel(T* d_input, T* d_output, int3 newdims, int3 olddims);
	__global__ void Iteration1Kernel(tcomplex* d_convft, tfloat* d_Fweight, tfloat* d_Fnewweight, uint elements);
	__global__ void Iteration2Kernel(tcomplex* d_convft, tfloat* d_Fnewweight, uint elements);
	__global__ void CorrectGriddingKernel(tfloat* d_volume, int dim, int oripadded);


	void d_ReconstructGridding(tcomplex* d_dataft, tfloat* d_weight, tfloat* d_reconstructed, int3 dimsori, int3 dimspadded, int paddingfactor, hipfftHandle pre_planforw, hipfftHandle pre_planback, int iterations, double blobradius, int bloborder, double blobalpha)
	{
		int3 dimsoripad = dimsori * paddingfactor;

		// Precalc blob values
		tfloat* d_precompblob;
		if (iterations > 0)
		{
			double radius = blobradius * paddingfactor;
			double alpha = blobalpha;
			int order = bloborder;
			int elements = 10000;
			double sampling = 0.5 / elements;
			tfloat* h_blobvalues = (tfloat*)malloc(elements * sizeof(tfloat));
			for (int i = 0; i < elements; i++)
				h_blobvalues[i] = kaiser_Fourier_value(i * sampling, radius, alpha, order);
			float blobnorm = h_blobvalues[0];
			for (int i = 0; i < elements; i++)
				h_blobvalues[i] /= blobnorm;
			tfloat* d_blobvalues = (tfloat*)CudaMallocFromHostArray(h_blobvalues, elements * sizeof(tfloat));

			hipMalloc((void**)&d_precompblob, Elements(dimsoripad) * sizeof(tfloat));
			int TpB = tmin(128, NextMultipleOf(dimsoripad.x, 32));
			dim3 grid = dim3(dimsoripad.y, dimsoripad.z);
			PrecomputeBlobKernel << <grid, TpB >> > (d_precompblob, dimsoripad.x, dimsoripad.x, d_blobvalues, (tfloat)sampling, elements);

			//d_WriteMRC(d_blobvalues, toInt3(elements, 1, 1), "d_blobvalues.mrc");
			//d_WriteMRC(d_precompblob, dimsoripad, "d_precompblob.mrc");

			hipFree(d_blobvalues);
			free(h_blobvalues);
		}

		int r_max = dimsori.x / 2;

		tcomplex* d_convft;
		hipMalloc((void**)&d_convft, ElementsFFT(dimsoripad) * sizeof(tcomplex));
		tfloat* d_conv;
		hipMalloc((void**)&d_conv, Elements(dimsoripad) * sizeof(tfloat));

		tfloat* d_Fweight;
		hipMalloc((void**)&d_Fweight, ElementsFFT(dimsoripad) * sizeof(tfloat));
		{
			int TpB = tmin(128, NextMultipleOf(dimsoripad.x, 32));
			dim3 grid = dim3(dimsoripad.y, dimsoripad.z);
			DecenterKernel<tfloat> <<<grid, TpB>>> (d_weight, d_Fweight, dimsori * paddingfactor, dimspadded);
		}
		//d_WriteMRC(d_Fweight, toInt3FFT(dimsoripad), "d_Fweight.mrc");

		// Fnewweight is initialized to 1 within r_max
		tfloat* d_Fnewweight = CudaMallocValueFilled(ElementsFFT(dimsoripad), (tfloat)1);
		d_SphereMaskFT(d_Fnewweight, d_Fnewweight, dimsoripad, r_max * paddingfactor);
		//d_WriteMRC(d_Fnewweight, toInt3FFT(dimsoripad), "d_Fnewweight.mrc");

		hipfftHandle planforw = pre_planforw, planback = pre_planback;
		if (pre_planforw <= NULL)
			planforw = d_FFTR2CGetPlan(3, dimsoripad);
		if (pre_planback <= NULL)
			planback = d_IFFTC2RGetPlan(3, dimsoripad);

		for (int i = 0; i < iterations; i++)
		{
			int TpB = 128;
			dim3 grid = dim3((ElementsFFT(dimsoripad) + TpB - 1) / TpB, 1, 1);
			Iteration1Kernel << <grid, TpB >> > (d_convft, d_Fweight, d_Fnewweight, ElementsFFT(dimsoripad));

			// Convolute with blob in real space
			d_IFFTC2R(d_convft, d_conv, &planback, dimsoripad);
			d_MultiplyByVector(d_conv, d_precompblob, d_conv, Elements(dimsoripad));
			d_FFTR2C(d_conv, d_convft, &planforw);

			Iteration2Kernel << <grid, TpB >> > (d_convft, d_Fnewweight, ElementsFFT(dimsoripad));

		}
		//d_MinOp(d_Fnewweight, 1e20f, d_Fnewweight, ElementsFFT(dimsoripad));
		//d_WriteMRC(d_Fnewweight, toInt3FFT(dimsoripad), "d_Fnewweight.mrc");

		{
			int TpB = tmin(128, NextMultipleOf(dimsoripad.x, 32));
			dim3 grid = dim3(dimsoripad.y, dimsoripad.z);
			DecenterKernel<tcomplex> << <grid, TpB >> > (d_dataft, d_convft, dimsori * paddingfactor, dimspadded);
		}

		if (iterations == 0)
		{
			//tcomplex* h_convft = (tcomplex*)MallocFromDeviceArray(d_convft, ElementsFFT(dimsoripad) * sizeof(tcomplex));
			//tfloat* h_Fweight = (tfloat*)MallocFromDeviceArray(d_Fweight, ElementsFFT(dimsoripad) * sizeof(tfloat));

			//for (size_t i = 0; i < ElementsFFT(dimsoripad); i++)
			//{
			//	if (abs(h_Fweight[i]) > 1e-4f)
			//	{
			//		h_convft[i] *= 1 / h_Fweight[i];
			//		//h_Fweight[i] *= 1 / h_Fweight[i];
			//	}
			//}

			//hipMemcpy(d_convft, h_convft, ElementsFFT(dimsoripad) * sizeof(tcomplex), hipMemcpyHostToDevice);
			////hipMemcpy(d_Fweight, h_Fweight, ElementsFFT(dimsoripad) * sizeof(tfloat), hipMemcpyHostToDevice);
			//free(h_convft);
			//free(h_Fweight);

			d_Abs(d_Fweight, d_Fweight, ElementsFFT(dimsoripad));
			d_MaxOp(d_Fweight, 1e-4f, d_Fweight, ElementsFFT(dimsoripad));
			d_ComplexDivideByVector(d_convft, d_Fweight, d_convft, ElementsFFT(dimsoripad), 1);
		}
		else
		{
			d_ComplexMultiplyByVector(d_convft, d_Fnewweight, d_convft, ElementsFFT(dimsoripad));
		}

		hipFree(d_Fweight);
		hipFree(d_Fnewweight);
		if (iterations > 0)
			hipFree(d_precompblob);

		tfloat3 decenter_shift[] = { tfloat3(dimsoripad.x / 2) };
		d_Shift(d_convft, d_convft, dimsoripad, decenter_shift);

		d_IFFTC2R(d_convft, d_conv, &planback);
		//d_WriteMRC(d_conv, dimsori * paddingfactor, "d_reconstructed.mrc");
		d_Pad(d_conv, d_reconstructed, dimsoripad, dimsori, T_PAD_MODE::T_PAD_VALUE, (tfloat)0);
		//d_RemapFullFFT2Full(d_reconstructed, d_reconstructed, dimsori);

		if (pre_planforw <= NULL)
			hipfftDestroy(planforw);
		if (pre_planback <= NULL)
			hipfftDestroy(planback);

		tfloat rf = r_max - 1;
		//d_SphereMask(d_reconstructed, d_reconstructed, dimsori, &rf, 3.0f, NULL);

		{
			int TpB = tmin(128, NextMultipleOf(dimsori.x, 32));
			dim3 grid = dim3(dimsori.y, dimsori.z);
			CorrectGriddingKernel <<<grid, TpB>>> (d_reconstructed, dimsori.x, dimsori.x * paddingfactor);
		}
		//d_DivideByScalar(d_reconstructed, d_reconstructed, Elements(dimsori), (tfloat)paddingfactor * paddingfactor * paddingfactor);
		//d_WriteMRC(d_reconstructed, dimsori, "d_reconstructed.mrc");

		d_MultiplyByScalar(d_reconstructed, d_reconstructed, Elements(dimsori), 1.0f / (paddingfactor * paddingfactor * paddingfactor * dimsori.x));

		hipFree(d_conv);
		hipFree(d_convft);
	}

	__global__ void PrecomputeBlobKernel(tfloat* d_precompblob, int paddim, int oripadded, tfloat* d_funcvals, tfloat funcsampling, int funcelements)
	{
		int z = blockIdx.y;
		int y = blockIdx.x;

		d_precompblob += (z * paddim + y) * paddim;

		int zp = z < paddim / 2 ? z : z - paddim;
		zp *= zp;
		int yp = y < paddim / 2 ? y : y - paddim;
		yp *= yp;

		for (int x = threadIdx.x; x < paddim; x += blockDim.x)
		{
			float xp = x < paddim / 2 ? x : x - paddim;
			float r = sqrt(xp * xp + yp + zp) / oripadded / funcsampling;

			d_precompblob[x] = d_funcvals[tmin(funcelements - 1, (int)r)];
		}
	}

	template<class T> __global__ void DecenterKernel(T* d_input, T* d_output, int3 newdims, int3 olddims)
	{
		int z = blockIdx.y;
		int y = blockIdx.x;
		
		float r = 0;
		int zp = z < newdims.z / 2 + 1 ? z : z - newdims.x;
		r += zp * zp;
		zp += olddims.z / 2;
		int yp = y < newdims.y / 2 + 1 ? y : y - newdims.x;
		r += yp * yp;
		yp += olddims.y / 2;

		for (int x = threadIdx.x; x < newdims.x / 2 + 1; x += blockDim.x)
		{
			int xp = x;
			float rr = r + xp * xp;
			float mask = rr < newdims.x * newdims.x / 4 ? 1 : 0;

			d_output[(z * newdims.y + y) * (newdims.x / 2 + 1) + x] = d_input[(zp * olddims.y + yp) * (olddims.x / 2 + 1) + xp] * mask;
		}
	}
	
	__global__ void Iteration1Kernel(tcomplex* d_convft, tfloat* d_Fweight, tfloat* d_Fnewweight, uint elements)
	{
		for (uint i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			d_convft[i] = make_hipComplex(d_Fweight[i] * d_Fnewweight[i], 0);
	}

	__global__ void Iteration2Kernel(tcomplex* d_convft, tfloat* d_Fnewweight, uint elements)
	{
		for (uint i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
		{
			//if (d_convft[i].x >= 0)
			{
				tfloat w = tmax(1e-4f, d_convft[i].x);
				d_Fnewweight[i] = tmin(1e20f, d_Fnewweight[i] / w);
			}
			/*else
			{
				tfloat w = tmin(-1e-6f, d_convft[i].x);
				d_Fnewweight[i] = tmin(1e20f, d_Fnewweight[i] / w);
			}*/
		}
	}

	__global__ void CorrectGriddingKernel(tfloat* d_volume, int dim, int oripadded)
	{
		int z = blockIdx.y;
		int y = blockIdx.x;

		d_volume += (z * dim + y) * dim;
		
		y -= dim / 2;
		y *= y;
		z -= dim / 2;
		z *= z;

		for (int x = threadIdx.x; x < dim; x += blockDim.x)
		{
			float xx = x - dim / 2;

			float r = sqrt(xx * xx + y + z);
			r /= oripadded;

			if (r > 0)
				d_volume[x] /= sinc(r) * sinc(r);
		}
	}
}