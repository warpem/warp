#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/CTF.cuh"
#include "gtom/include/DeviceFunctions.cuh"
#include "gtom/include/CubicInterp.cuh"
#include "gtom/include/Generics.cuh"
#include "gtom/include/Helper.cuh"
#include "gtom/include/ImageManipulation.cuh"
#include "gtom/include/Masking.cuh"


namespace gtom
{
    __global__ void DistortKernel(cudaTex t_image, int2 dimsinput, tfloat* d_output, uint dimoutput, float2* d_offsets, glm::mat2* d_transforms);
    __global__ void WarpKernel(cudaTex t_image, tfloat* d_output, int2 dims, int2 dimswarp, cudaTex t_warpx, cudaTex t_warpy);

    //////////////////////////
    //Corrects for beam tilt//
    //////////////////////////

    void d_DistortImages(tfloat* d_input, int2 dimsinput, tfloat* d_output, int2 dimsoutput, float2* h_offsets, float* h_rotations, float3* h_scales, uint batch)
    {
        hipArray_t a_image;
        cudaTex t_image;

        {
            d_BindTextureTo3DArray(d_input, a_image, t_image, toInt3(dimsinput.x, dimsinput.y, batch), hipFilterModePoint, false);
        }

        glm::mat2* h_transforms = (glm::mat2*)malloc(batch * sizeof(glm::mat2));
        for (int i = 0; i < batch; i++)
            h_transforms[i] = Matrix2Rotation(h_rotations[i]) * Matrix2Rotation(h_scales[i].z) * Matrix2Scale(tfloat2(1.0f / h_scales[i].x, 1.0f / h_scales[i].y)) * Matrix2Rotation(-h_scales[i].z);

        glm::mat2* d_transforms = (glm::mat2*)CudaMallocFromHostArray(h_transforms, batch * sizeof(glm::mat2));
        free(h_transforms);

        float2* d_offsets = (float2*)CudaMallocFromHostArray(h_offsets, batch * sizeof(float2));

        dim3 grid = dim3(tmin(32768, (Elements2(dimsoutput) + 127) / 128), batch, 1);
        DistortKernel << <grid, 128 >> > (t_image, dimsinput, d_output, dimsoutput.x, d_offsets, d_transforms);

        hipFree(d_offsets);
        hipFree(d_transforms);

        {
            hipDestroyTextureObject(t_image);
            hipFreeArray(a_image);
        }
    }

	void d_DistortImages(tfloat* d_input, int2 dimsinput, tfloat* d_output, int2 dimsoutput, float2* h_offsets, float4* h_distortions, uint batch)
	{
		hipArray_t a_image;
		cudaTex t_image;

		{
			d_BindTextureTo3DArray(d_input, a_image, t_image, toInt3(dimsinput.x, dimsinput.y, batch), hipFilterModePoint, false);
		}

		glm::mat2* h_transforms = (glm::mat2*)malloc(batch * sizeof(glm::mat2));
		for (int i = 0; i < batch; i++)
			h_transforms[i] = glm::mat2(h_distortions[i].x, h_distortions[i].y, h_distortions[i].z, h_distortions[i].w);

		glm::mat2 * d_transforms = (glm::mat2*)CudaMallocFromHostArray(h_transforms, batch * sizeof(glm::mat2));
		free(h_transforms);

		float2 * d_offsets = (float2*)CudaMallocFromHostArray(h_offsets, batch * sizeof(float2));

		dim3 grid = dim3(tmin(32768, (Elements2(dimsoutput) + 127) / 128), batch, 1);
		DistortKernel << <grid, 128 >> > (t_image, dimsinput, d_output, dimsoutput.x, d_offsets, d_transforms);

		hipFree(d_offsets);
		hipFree(d_transforms);

		{
			hipDestroyTextureObject(t_image);
			hipFreeArray(a_image);
		}
	}

    void d_WarpImage(tfloat* d_input, tfloat* d_output, int2 dims, tfloat* h_warpx, tfloat* h_warpy, int2 dimswarp, hipArray_t a_input)
    {
        hipArray_t a_image, a_warpx, a_warpy;
        cudaTex t_image, t_warpx, t_warpy;

        tfloat* d_warpx = (tfloat*)CudaMallocFromHostArray(h_warpx, Elements2(dimswarp) * sizeof(tfloat));
        tfloat* d_warpy = (tfloat*)CudaMallocFromHostArray(h_warpy, Elements2(dimswarp) * sizeof(tfloat));

        {
			if (a_input == NULL)
			{
				d_BindTextureToArray(d_input, a_image, t_image, dims * 1, hipFilterModeLinear, false);
			}
			else
			{
				d_CopyToArray(d_input, a_input, dims.x, dims.y, 1, sizeof(tfloat), hipMemcpyDeviceToDevice);
				d_BindTextureToArray(a_input, t_image, dims * 1, hipFilterModeLinear, false);
			}

            d_BindTextureToArray(d_warpx, a_warpx, t_warpx, dimswarp, hipFilterModeLinear, false);
            d_BindTextureToArray(d_warpy, a_warpy, t_warpy, dimswarp, hipFilterModeLinear, false);
        }

        dim3 grid = dim3(tmin(32768, (Elements2(dims) + 127) / 128), 1, 1);
        WarpKernel << <grid, 128 >> > (t_image, d_output, dims, dimswarp, t_warpx, t_warpy);

        hipFree(d_warpx);
        hipFree(d_warpy);

        hipDestroyTextureObject(t_image);
		if (a_input == NULL)
			hipFreeArray(a_image);

        hipDestroyTextureObject(t_warpx);
        hipFreeArray(a_warpx);
        hipDestroyTextureObject(t_warpy);
        hipFreeArray(a_warpy);
    }

    __global__ void DistortKernel(cudaTex t_image, int2 dimsinput, tfloat* d_output, uint dimoutput, float2* d_offsets, glm::mat2* d_transforms)
    {
        d_output += dimoutput * dimoutput * blockIdx.y;
        float zcoord = blockIdx.y + 0.5f;
        int2 inputcenter = make_int2(dimsinput.x / 2, dimsinput.y / 2);
        int outputcenter = dimoutput / 2;

        glm::mat2 transform = d_transforms[blockIdx.y];
        float2 offset = d_offsets[blockIdx.y];

        for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < dimoutput * dimoutput; id += gridDim.x * blockDim.x)
        {
            uint idx = id % dimoutput;
            uint idy = id / dimoutput;

            int posx = (int)idx - outputcenter;
            int posy = (int)idy - outputcenter;

            glm::vec2 pos = transform * glm::vec2(posx, posy);

            pos.x += inputcenter.x - offset.x;
            pos.y += inputcenter.y - offset.y;


            //tfloat val = tex3D<tfloat>(t_image, pos.x + 0.5f, pos.y + 0.5f, zcoord);
            tfloat val = 0, weights = 0;

			for (int y = -8; y <= 8; y++)
			{
				float yy = floor(pos.y) + y;
				float sincy = sinc(pos.y - yy) * sinc((pos.y - yy) / 8);
				float yy2 = pos.y - yy;
				yy2 *= yy2;
				yy += 0.5f;

				for (int x = -8; x <= 8; x++)
				{
					float xx = floor(pos.x) + x;
					float sincx = sinc(pos.x - xx) * sinc((pos.x - xx) / 8);
					float xx2 = pos.x - xx;
					xx2 *= xx2;
					/*float r2 = xx2 + yy2;

					if (r2 > 64)
					continue;

					float hanning = 1.0f + cos(PI * sqrt(r2) / 8);*/	// Let's try Lanczos instead

					tfloat weight = sincy * sincx;
					val += tex3D<tfloat>(t_image, xx + 0.5f, yy, zcoord) * weight;// *hanning;
					weights += weight;
				}
			}

            d_output[id] = val / weights;
        }
    }

    __global__ void WarpKernel(cudaTex t_image, tfloat* d_output, int2 dims, int2 dimswarp, cudaTex t_warpx, cudaTex t_warpy)
    {
        for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < Elements2(dims); id += gridDim.x * blockDim.x)
        {
            uint idx = id % (uint)dims.x;
            uint idy = id / (uint)dims.x;

            int posx = (int)idx;
            int posy = (int)idy;

            float2 warppos = make_float2((float)posx / (dims.x - 1) * (dimswarp.x - 1), (float)posy / (dims.y - 1) * (dimswarp.y - 1));
            float2 pos = make_float2(posx - tex2D<tfloat>(t_warpx, warppos.x, warppos.y),
                                     posy - tex2D<tfloat>(t_warpy, warppos.x, warppos.y));

			//tfloat val = 0, weights = 0;

   //         for (int y = -6; y <= 6; y++)
   //         {
   //             float yy = floor(pos.y) + y;
   //             float sincy = sinc(pos.y - yy) * sinc((pos.y - yy) / 6);
   //             float yy2 = pos.y - yy;
   //             yy2 *= yy2;
   //             yy += 0.5f;

   //             for (int x = -6; x <= 6; x++)
   //             {
   //                 float xx = floor(pos.x) + x;
   //                 float sincx = sinc(pos.x - xx) * sinc((pos.x - xx) / 6);
   //                 float xx2 = pos.x - xx;
   //                 xx2 *= xx2;
   //                 /*float r2 = xx2 + yy2;

   //                 if (r2 > 64)
   //                     continue;

   //                 float hanning = 1.0f + cos(PI * sqrt(r2) / 8);*/	// Let's try Lanczos instead

			//		tfloat weight = sincy * sincx;
			//		val += tex2D<tfloat>(t_image, xx + 0.5f, yy) * weight;// *hanning;
			//		weights += weight;
   //             }
   //         }

   //         d_output[id] = val / weights;

			//pos *= 1;
            d_output[id] = cubicTex2D(t_image, pos.x + 0.5f, pos.y + 0.5f);
        }
    }
}